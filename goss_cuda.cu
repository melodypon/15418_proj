#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <driver_functions.h>
#include <thrust/sort.h>
#include <math.h>
#include <algorithm>
#include <numeric>
#include "timing.h"

#define ITEM_PER_THREAD 8

__global__ void gossKernel(int NumberCount, int topN, int randN, float* predictions, float* train, float* gradients, int* usedSet) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < NumberCount) {
        for (int i = index * ITEM_PER_THREAD; i < (index + 1) * ITEM_PER_THREAD; i++) {
            if (i >= NumberCount) {
                break;
            }
            gradients[i] = std::fabs(predictions[i] - train[i]);
        }
    }
}

__global__ void getUsedSet(int* usedSet, int* indices, int topN, int randN) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = index * ITEM_PER_THREAD; i < (index + 1) * ITEM_PER_THREAD; i++) {
        if (i > topN + randN) {
            break;
        }
        usedSet[i] = indices[i];
    }
}

void gossCuda(int NumberCount, int topN, int randN, float* predictions, float* train, int* usedSet) {
    const int threadsPerBlock = 512;
    const int blocks = (NumberCount / ITEM_PER_THREAD + threadsPerBlock - 1) / threadsPerBlock;

    float* gradients = (float*)calloc(NumberCount, sizeof(float));

    float* device_pred;
    float* device_train;
    float* device_grad;
    int* device_indices;
    int* device_usedset;

    hipMalloc(&device_pred, sizeof(float) * NumberCount);
    hipMalloc(&device_train, sizeof(float) * NumberCount);
    hipMalloc(&device_grad, sizeof(float) * NumberCount);
    hipMalloc(&device_usedset, sizeof(int) * (topN + randN));

    hipMemcpy(device_pred, predictions, sizeof(float) * NumberCount, hipMemcpyHostToDevice);
    hipMemcpy(device_train, train, sizeof(float) * NumberCount, hipMemcpyHostToDevice);

    Timer timer1;
    gossKernel<<<blocks, threadsPerBlock>>>(NumberCount, topN, randN, device_pred, device_train, device_grad, device_usedset);
    hipDeviceSynchronize();
    double t1 = timer1.elapsed();

    Timer timer2;
    std::vector<int> indices(NumberCount);
    iota(indices.begin(), indices.end(), 0);
    hipMemcpy(gradients, device_grad, sizeof(float) * NumberCount, hipMemcpyDeviceToHost);
    thrust::sort_by_key(indices.data(), indices.data() + NumberCount, gradients);
    double t2 = timer2.elapsed();
    
    hiprandGenerator_t gen;
    float* dev_rand;
    float* rand = (float*)calloc(NumberCount - topN, sizeof(float));
    hipMalloc(&dev_rand, (NumberCount - topN) * sizeof(float));
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandGenerateUniform(gen, dev_rand, (NumberCount - topN));
    hipMemcpy(rand, dev_rand, (NumberCount - topN) * sizeof(float), hipMemcpyDeviceToHost);

    Timer timer3;
    thrust::sort_by_key(indices.data() + topN, indices.data() + NumberCount, rand);
    double t3 = timer3.elapsed();

    Timer timer4;
    getUsedSet<<<blocks, threadsPerBlock>>>(device_usedset, device_indices, topN, randN);
    double t4 = timer4.elapsed();

    hipMemcpy(usedSet, device_usedset, sizeof(int) * (topN + randN), hipMemcpyDeviceToHost);

    printf("TOTAL TIME  : %.6fs\n", t1 + t2 + t3 + t4);
    printf("Compute grad: %.6fs\n", t1);
    printf("Sort by grad: %.6fs\n", t2);
    printf("Sampling    : %.6fs\n", t3);
    printf("New dataset : %.6fs\n", t4);
}