#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <driver_functions.h>
#include <thrust/sort.h>
#include <math.h>
#include <algorithm>
#include <numeric>
#include <random>
#include "timing.h"

#define ITEM_PER_THREAD 8

__global__ void gossKernel(int NumberCount, int topN, int randN, float* predictions, float* train, float* gradients, int* usedSet) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < NumberCount) {
        for (int i = index * ITEM_PER_THREAD; i < (index + 1) * ITEM_PER_THREAD; i++) {
            if (i >= NumberCount) {
                break;
            }
            gradients[i] = std::fabs(predictions[i] - train[i]);
        }
    }
}

/*__global__ void getUsedSet(int* usedSet, int* indices, int topN, int randN) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = index * ITEM_PER_THREAD; i < (index + 1) * ITEM_PER_THREAD; i++) {
        if (i > topN + randN) {
            break;
        }
        usedSet[i] = indices[i];
    }
}*/

void getUsedSet(int* usedSet, std::vector<int> &indices, std::vector<int> &randSet, int topN, int randN) {
    for (int i  = 0; i < topN; i++) {
        usedSet[i] = indices[i];
    }
    for (int i = topN; i < randN + topN; i++) {
        usedSet[i] = randSet[i - topN];
    }
}

void gossCuda(int NumberCount, int topN, int randN, float* predictions, float* train, int* usedSet) {
    const int threadsPerBlock = 512;
    const int blocks = (NumberCount / ITEM_PER_THREAD + threadsPerBlock - 1) / threadsPerBlock;

    float* gradients = (float*)calloc(NumberCount, sizeof(float));

    float* device_pred;
    float* device_train;
    float* device_grad;
    int* device_indices;
    int* device_usedset;

    Timer timer5;
    hipMalloc(&device_pred, sizeof(float) * NumberCount);
    hipMalloc(&device_train, sizeof(float) * NumberCount);
    hipMalloc(&device_grad, sizeof(float) * NumberCount);
    hipMalloc(&device_usedset, sizeof(int) * (topN + randN));

    hipMemcpy(device_pred, predictions, sizeof(float) * NumberCount, hipMemcpyHostToDevice);
    hipMemcpy(device_train, train, sizeof(float) * NumberCount, hipMemcpyHostToDevice);
    double t5 = timer5.elapsed();

    Timer timer1;
    gossKernel<<<blocks, threadsPerBlock>>>(NumberCount, topN, randN, device_pred, device_train, device_grad, device_usedset);
    hipDeviceSynchronize();
    double t1 = timer1.elapsed();

    Timer timer2;
    std::vector<int> indices(NumberCount);
    iota(indices.begin(), indices.end(), 0);
    hipMemcpy(gradients, device_grad, sizeof(float) * NumberCount, hipMemcpyDeviceToHost);
    thrust::sort_by_key(indices.data(), indices.data() + NumberCount, gradients);
    double t2 = timer2.elapsed();

    Timer timer3;
    std::vector<int> randSet;
    std::sample(indices.begin() + topN, indices.end(), std::back_inserter(randSet), randN, std::mt19937{std::random_device{}()});
    double t3 = timer3.elapsed();

    Timer timer4;
    getUsedSet(usedSet, indices, randSet, topN, randN);
    double t4 = timer4.elapsed();
    
    /* Timer timer7;
    hiprandGenerator_t gen;
    float* dev_rand;
    // float* rand = (float*)calloc(NumberCount - topN, sizeof(float));
    hipMalloc(&dev_rand, (NumberCount - topN) * sizeof(float));
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandGenerateUniform(gen, dev_rand, (NumberCount - topN));
    // hipMemcpy(rand, dev_rand, (NumberCount - topN) * sizeof(float), hipMemcpyDeviceToHost);
    double t7 = timer7.elapsed();

    Timer timer3;
    thrust::sort_by_key(indices.data() + topN, indices.data() + NumberCount, rand);
    double t3 = timer3.elapsed();

    Timer timer4;
    getUsedSet<<<blocks, threadsPerBlock>>>(device_usedset, device_indices, topN, randN);
    double t4 = timer4.elapsed();

    Timer timer6;
    hipMemcpy(usedSet, device_usedset, sizeof(int) * (topN + randN), hipMemcpyDeviceToHost);
    double t6 = timer6.elapsed(); */

    printf("TOTAL TIME  : %.6fs\n", t1 + t2 + t3 + t4 + t5);
    printf("Compute grad: %.6fs\n", t1);
    printf("Sort by grad: %.6fs\n", t2);
    printf("Sampling    : %.6fs\n", t3);
    printf("New dataset : %.6fs\n", t4);
    printf("Datamovement : %.6fs\n", t5);
}